#include "hip/hip_runtime.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <numeric>
#include <algorithm>
#include <chrono>
#include <array>
#include <thread>
#include <atomic>

struct color {float r, g, b, a;};
struct rawcolor{ unsigned char r, g, b, a; };

__global__ void TEX_identity_trafo(unsigned int* out, hipTextureObject_t texObjInput, const unsigned int* histo, int W, int H){
    unsigned int x0 = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y0 = blockIdx.y*blockDim.y + threadIdx.y;

    // calc intensity (f_ij):
    float4 pixel = tex2D<float4>(texObjInput, x0, y0);
    unsigned int fij = (unsigned int)(((0.3 * pixel.x) + (0.59 * pixel.y) + (0.11 * pixel.z))*255);

    // most csak identitás trafó (nem használ histo):
    out[y0*W+x0] = fij;
}


int main()
{
    static const std::string input_filename   = "input.jpg";
    static const std::string output_filename1 = "cpu_out1.jpg";
    static const std::string output_filename2 = "gpu_out1.jpg";
    static const std::string output_filename3 = "gpu_out2.jpg";

    static const int block_size = 16;
    //int nBlocksW = 0; //number of blocks horizontally, not used now
    int nBlocksH = 0; //number of blocks vertically
    
    int w = 0;//width
    int h = 0;//height
    int ch = 0;//number of components

    rawcolor* data0 = reinterpret_cast<rawcolor*>(stbi_load(input_filename.c_str(), &w, &h, &ch, 4));
    if(!data0)
    {
        std::cout << "Error: could not open input file: " << input_filename << "\n";
        return -1;
    }
    else
    {
        //nBlocksW = w / block_size; //not used now
        nBlocksH = h / block_size;
        std::cout << "Image (" << input_filename << ") opened successfully. Width x Height x Components = " << w << " x " << h << " x " << ch << "\n";
    }

    hipError_t err = hipSuccess;
    std::vector<unsigned int> hostOutput(w*h);
    std::vector<unsigned int> hostOutputShared(w*h);
    unsigned int*  hPartials = nullptr;
    unsigned int*  hOutput   = nullptr;
    unsigned int*  pImageOut = nullptr;

    float dt2 = 0.0f;
    {
        err = hipMalloc( (void**)&hPartials, nBlocksH*256*sizeof(unsigned int) );
        if( err != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }

        err = hipMemset(hPartials, 0, nBlocksH*256*sizeof(unsigned int) );
        if( err != hipSuccess){ std::cout << "Error setting memory to zero: " << hipGetErrorString(err) << "\n"; return -1; }
        
        err = hipMalloc( (void**)&hOutput, 256*sizeof(unsigned int) );
        if( err != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }

        err = hipMalloc( (void**)&pImageOut, w*h*sizeof(unsigned int) );
        if( err != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
        err = hipMemset(pImageOut, 0, w*h*sizeof(unsigned int) );
        if( err != hipSuccess){ std::cout << "Error setting memory to zero: " << hipGetErrorString(err) << "\n"; return -1; }

        // Initialize texture
        // Ehhez transform data to color:
        std::vector<color> input(w*h);
        std::transform(data0, data0+w*h, input.begin(), 
        [](rawcolor c){ return color{c.r/255.0f, c.g/255.0f, c.b/255.0f, c.a/255.0f}; }
        );

        //Kell egy hipArray ami az adatot tárolja
        //Kell egy hipResourceDesc ami jellemzi
        //Kell hipTextureDesc ami jellemzi a textúrát
        //Creation: hipCreateTextureObject fv -> hipTextureObject_t objektum, át lehet adni kernelnek
        
        // Cuda Array:
        hipChannelFormatDesc channelDescInput = hipCreateChannelDesc(32,32,32,32, hipChannelFormatKindFloat);
        hipArray* aInput;

        // Malloc and load to device:
        err = hipMallocArray(&aInput,&channelDescInput, w, h);
        if( err != hipSuccess){ std::cout << "Error allocating CUDA memory (tex): " << hipGetErrorString(err) << "\n"; return -1; }
        err = hipMemcpyToArray(aInput, 0, 0, input.data(), w*h*sizeof(color), hipMemcpyHostToDevice);
        if( err != hipSuccess){ std::cout << "Error copying memory to device (tex): " << hipGetErrorString(err) << "\n"; return -1; }

        // hipResourceDesc írja le, hogy hipArray-ben van a cucc:
        hipResourceDesc resdescInput{}; // 0-ra inicializálva
        resdescInput.resType = hipResourceTypeArray; //megmondja hogy array van
        resdescInput.res.array.array = aInput; // pointer rá

        // hipTextureDesc jellemzi a textúrát:
        hipTextureDesc texDesc{}; // 0 init
        texDesc.addressMode[0] = hipAddressModeClamp; // mi a van a htáron: az utolsó szín: Clamp
        texDesc.addressMode[1] = hipAddressModeClamp;
        texDesc.filterMode = hipFilterModeLinear; //lineáris szűrés
        texDesc.readMode = hipReadModeElementType; // nincs adatkonv. float-ra
        texDesc.normalizedCoords = 0; // [0.0, texture_size] a tartomány

        // Mindent összekapcs:
        hipTextureObject_t texObjInput = 0;
        err = hipCreateTextureObject(&texObjInput, &resdescInput, &texDesc, nullptr); 
        if( err != hipSuccess){ std::cout << "Error creating texture object: " << hipGetErrorString(err) << "\n"; return -1; }

        // KERNEL:
        {
            dim3 dimGrid( w / block_size, h / block_size + 1 );
            dim3 dimBlock( block_size, block_size );
            TEX_identity_trafo<<<dimGrid, dimBlock>>>(pImageOut, texObjInput, hOutput, w, h);
            err = hipGetLastError();
            if (err != hipSuccess){ std::cout << "CUDA error in third kernel call: " << hipGetErrorString(err) << "\n"; return -1; }
        }
        hipDeviceSynchronize();

        // eredmény visszamásolása gazda oldalra
        err = hipMemcpy( hostOutputShared.data(), pImageOut, w*h*sizeof(unsigned int), hipMemcpyDeviceToHost );
        if( err != hipSuccess){ std::cout << "Error copying memory to host at end: " << hipGetErrorString(err) << "\n"; return -1; }
    }


    stbi_image_free(data0);
    err = hipFree( hPartials );
    if( err != hipSuccess){ std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFree( hOutput );
    if( err != hipSuccess){ std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFree( pImageOut );
    if( err != hipSuccess){ std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }





    /* CONVERT WRITE & FINISH */
    auto convert_and_write = [w, h](std::string const& filename, std::vector<unsigned int> &image ){
        std::vector<rawcolor> data(w*h);
        for(int i=0; i< data.size(); i++){
            data[i].r = image[i];
            data[i].g = image[i];
            data[i].b = image[i];
            data[i].a = 255;
        }
        std::vector<rawcolor> tmp(w*h);
        std::transform(data.cbegin(), data.cend(), tmp.begin(),
            [](rawcolor c){ return rawcolor{   (unsigned char)(c.r),
                                            (unsigned char)(c.g),
                                            (unsigned char)(c.b),
                                            (unsigned char)(c.a) }; } );

        int res = stbi_write_jpg(filename.c_str(), w, h, 4, tmp.data(), 40);
        if(res == 0)
        {
            std::cout << "Error writing output to file " << filename << "\n";
        }else{ std::cout << "Output written to file " << filename << "\n"; }
    };

    convert_and_write(output_filename3, hostOutputShared);
}

